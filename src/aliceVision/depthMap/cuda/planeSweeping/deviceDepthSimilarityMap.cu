#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2022 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#include "deviceDepthSimilarityMap.hpp"
#include "deviceDepthSimilarityMapKernels.cuh"

#include <aliceVision/depthMap/cuda/host/divUp.hpp>

#include <utility>

namespace aliceVision {
namespace depthMap {

__host__ void cuda_depthSimMapCopyDepthOnly(CudaDeviceMemoryPitched<float2, 2>& out_depthSimMap_dmp,
                                            const CudaDeviceMemoryPitched<float2, 2>& in_depthSimMap_dmp,
                                            float defaultSim, 
                                            hipStream_t stream)
{
    const CudaSize<2>& depthSimMapSize = out_depthSimMap_dmp.getSize();

    const int blockSize = 16;
    const dim3 block(blockSize, blockSize, 1);
    const dim3 grid(divUp(depthSimMapSize.x(), blockSize), divUp(depthSimMapSize.y(), blockSize), 1);

    depthSimMapCopyDepthOnly_kernel<<<grid, block, 0, stream>>>(
      out_depthSimMap_dmp.getBuffer(), 
      out_depthSimMap_dmp.getPitch(), 
      in_depthSimMap_dmp.getBuffer(), 
      in_depthSimMap_dmp.getPitch(),
      depthSimMapSize.x(),
      depthSimMapSize.y(),
      defaultSim);

    CHECK_CUDA_ERROR();
}

__host__ void cuda_normalMapUpscale(CudaDeviceMemoryPitched<float3, 2>& out_upscaledMap_dmp,
                                    const CudaDeviceMemoryPitched<float3, 2>& in_map_dmp,
                                    const ROI& roi,
                                    hipStream_t stream)
{
    const CudaSize<2>& out_mapSize = out_upscaledMap_dmp.getSize();
    const CudaSize<2>& in_mapSize = in_map_dmp.getSize();

    const float ratio = float(in_mapSize.x()) / float(out_mapSize.x());

    const int blockSize = 16;
    const dim3 block(blockSize, blockSize, 1);
    const dim3 grid(divUp(roi.width(), blockSize), divUp(roi.height(), blockSize), 1);

    mapUpscale_kernel<float3><<<grid, block, 0, stream>>>(
      out_upscaledMap_dmp.getBuffer(),
      out_upscaledMap_dmp.getPitch(),
      in_map_dmp.getBuffer(),
      in_map_dmp.getPitch(),
      ratio,
      roi);

    CHECK_CUDA_ERROR();
}

__host__ void cuda_computeSgmUpscaledDepthPixSizeMap(CudaDeviceMemoryPitched<float2, 2>& out_upscaledDepthPixSizeMap_dmp,
                                                     const CudaDeviceMemoryPitched<float2, 2>& in_sgmDepthThiknessMap_dmp,
                                                     const DeviceCamera& rcDeviceCamera,
                                                     const RefineParams& refineParams,
                                                     const ROI& roi,
                                                     hipStream_t stream)
{
    const CudaSize<2>& out_mapSize = out_upscaledDepthPixSizeMap_dmp.getSize();
    const CudaSize<2>& in_mapSize = in_sgmDepthThiknessMap_dmp.getSize();

    const float ratio = float(in_mapSize.x()) / float(out_mapSize.x());

    const int blockSize = 16;
    const dim3 block(blockSize, blockSize, 1);
    const dim3 grid(divUp(roi.width(), blockSize), divUp(roi.height(), blockSize), 1);

    if(refineParams.interpolateMiddleDepth)
    {
        computeSgmUpscaledDepthPixSizeMap_bilinear_kernel<<<grid, block, 0, stream>>>(
          rcDeviceCamera.getTextureObject(),
          out_upscaledDepthPixSizeMap_dmp.getBuffer(),
          out_upscaledDepthPixSizeMap_dmp.getPitch(),
          in_sgmDepthThiknessMap_dmp.getBuffer(),
          in_sgmDepthThiknessMap_dmp.getPitch(),
          refineParams.stepXY,
          refineParams.halfNbDepths,
          ratio,
          roi);
    }
    else
    {
        computeSgmUpscaledDepthPixSizeMap_nearestNeighbor_kernel<<<grid, block, 0, stream>>>(
          rcDeviceCamera.getTextureObject(),
          out_upscaledDepthPixSizeMap_dmp.getBuffer(),
          out_upscaledDepthPixSizeMap_dmp.getPitch(),
          in_sgmDepthThiknessMap_dmp.getBuffer(),
          in_sgmDepthThiknessMap_dmp.getPitch(),
          refineParams.stepXY,
          refineParams.halfNbDepths,
          ratio,
          roi);
    }

    CHECK_CUDA_ERROR();
}

__host__ void cuda_depthSimMapComputeNormal(CudaDeviceMemoryPitched<float3, 2>& out_normalMap_dmp,
                                            const CudaDeviceMemoryPitched<float2, 2>& in_depthSimMap_dmp,
                                            const DeviceCamera& rcDeviceCamera, 
                                            const int stepXY,
                                            const ROI& roi,
                                            hipStream_t stream)
{
    const int wsh = 4;

    const dim3 block(8, 8, 1);
    const dim3 grid(divUp(roi.width(), block.x), divUp(roi.height(), block.y), 1);

    depthSimMapComputeNormal_kernel<<<grid, block, 0, stream>>>(
      rcDeviceCamera.getDeviceCamId(),
      out_normalMap_dmp.getBuffer(),
      out_normalMap_dmp.getPitch(),
      in_depthSimMap_dmp.getBuffer(),
      in_depthSimMap_dmp.getPitch(),
      wsh,
      stepXY,
      roi);

    CHECK_CUDA_ERROR();
}

__host__ void cuda_depthSimMapOptimizeGradientDescent(CudaDeviceMemoryPitched<float2, 2>& out_optimizeDepthSimMap_dmp,
                                                      CudaDeviceMemoryPitched<float, 2>& inout_imgVariance_dmp,
                                                      CudaDeviceMemoryPitched<float, 2>& inout_tmpOptDepthMap_dmp,
                                                      const CudaDeviceMemoryPitched<float2, 2>& in_sgmDepthPixSizeMap_dmp,
                                                      const CudaDeviceMemoryPitched<float2, 2>& in_refineDepthSimMap_dmp,
                                                      const DeviceCamera& rcDeviceCamera, 
                                                      const RefineParams& refineParams,
                                                      const ROI& roi,
                                                      hipStream_t stream)
{
    // initialize depth/sim map optimized with SGM depth/pixSize map
    out_optimizeDepthSimMap_dmp.copyFrom(in_sgmDepthPixSizeMap_dmp, stream);

    {
        // setup block and grid
        const dim3 lblock(32, 2, 1);
        const dim3 lgrid(divUp(roi.width(), lblock.x), divUp(roi.height(), lblock.y), 1);

        optimize_varLofLABtoW_kernel<<<lgrid, lblock, 0, stream>>>(
            rcDeviceCamera.getTextureObject(), 
            inout_imgVariance_dmp.getBuffer(), 
            inout_imgVariance_dmp.getPitch(),
            refineParams.stepXY,
            roi);
    }

    CudaTexture<float> imgVarianceTex(inout_imgVariance_dmp);
    CudaTexture<float> depthTex(inout_tmpOptDepthMap_dmp);

    // setup block and grid
    const int blockSize = 16;
    const dim3 block(blockSize, blockSize, 1);
    const dim3 grid(divUp(roi.width(), blockSize), divUp(roi.height(), blockSize), 1);

    for(int iter = 0; iter < refineParams.optimizationNbIterations; ++iter) // default nb iterations is 100
    {
        // copy depths values from out_depthSimMapOptimized_dmp to inout_tmpOptDepthMap_dmp
        optimize_getOptDeptMapFromOptDepthSimMap_kernel<<<grid, block, 0, stream>>>(
            inout_tmpOptDepthMap_dmp.getBuffer(), 
            inout_tmpOptDepthMap_dmp.getPitch(), 
            out_optimizeDepthSimMap_dmp.getBuffer(), // initialized with SGM depth/sim map
            out_optimizeDepthSimMap_dmp.getPitch(),
            roi);

        // adjust depth/sim by using previously computed depths
        optimize_depthSimMap_kernel<<<grid, block, 0, stream>>>(
            rcDeviceCamera.getDeviceCamId(), 
            imgVarianceTex.textureObj,
            depthTex.textureObj, 
            out_optimizeDepthSimMap_dmp.getBuffer(),
            out_optimizeDepthSimMap_dmp.getPitch(),
            in_sgmDepthPixSizeMap_dmp.getBuffer(),
            in_sgmDepthPixSizeMap_dmp.getPitch(),
            in_refineDepthSimMap_dmp.getBuffer(),
            in_refineDepthSimMap_dmp.getPitch(),
            iter, 
            roi);
    }

    CHECK_CUDA_ERROR();
}

} // namespace depthMap
} // namespace aliceVision
